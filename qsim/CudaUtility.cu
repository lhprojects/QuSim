#include "CudaUtility.h"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_ptr.h>

struct my_multiplies {
	typedef hipDoubleComplex first_argument_type;
	typedef hipDoubleComplex second_argument_type;
	typedef hipDoubleComplex result_type;
	__host__ __device__ hipDoubleComplex operator()(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) const
	{
		return hipCmul(lhs, rhs);
	}
};

struct my_scale {
	typedef hipDoubleComplex first_argument_type;
	typedef hipDoubleComplex second_argument_type;
	typedef hipDoubleComplex result_type;

	my_scale(double m) : scale(m) { }
	double scale;
	__host__ __device__ hipDoubleComplex operator()(const hipDoubleComplex &lhs) const
	{
		return make_hipDoubleComplex(hipCreal(lhs)*scale, hipCimag(lhs)*scale);
	}
};

hipError_t cudaProduct(hipDoubleComplex *out, hipDoubleComplex * f2, hipDoubleComplex * f1, size_t n)
{
	thrust::device_ptr<hipDoubleComplex> d_f1 = thrust::device_pointer_cast(f1);
	thrust::device_ptr<hipDoubleComplex> d_f1_e = d_f1 + n;
	thrust::device_ptr<hipDoubleComplex> d_f2 = thrust::device_pointer_cast(f2);
	thrust::device_ptr<hipDoubleComplex> d_out = thrust::device_pointer_cast(out);

	thrust::transform(d_f1, d_f1_e, d_f2, d_out, my_multiplies());
	return hipSuccess;
}

hipError_t cudaScale(hipDoubleComplex *out, hipDoubleComplex *in, double alpha, size_t n)
{
	thrust::device_ptr<hipDoubleComplex> d_in = thrust::device_pointer_cast(in);
	thrust::device_ptr<hipDoubleComplex> d_in_e = d_in + n;
	thrust::device_ptr<hipDoubleComplex> d_out = thrust::device_pointer_cast(out);

	thrust::transform(d_in, d_in_e, d_out, my_scale(alpha));
	return hipSuccess;
}
