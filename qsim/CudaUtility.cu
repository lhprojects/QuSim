#include "hip/hip_runtime.h"
#include "CudaUtility.h"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_ptr.h>

struct my_multiplies {
	typedef hipDoubleComplex first_argument_type;
	typedef hipDoubleComplex second_argument_type;
	typedef hipDoubleComplex result_type;
	__host__ __device__ hipDoubleComplex operator()(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) const
	{
		return hipCmul(lhs, rhs);
	}
};

struct my_scale {
	typedef hipDoubleComplex first_argument_type;
	typedef hipDoubleComplex second_argument_type;
	typedef hipDoubleComplex result_type;

	my_scale(double m) : scale(m) { }
	double scale;
	__host__ __device__ hipDoubleComplex operator()(const hipDoubleComplex &lhs) const
	{
		return make_hipDoubleComplex(hipCreal(lhs)*scale, hipCimag(lhs)*scale);
	}
};

#if 0
__global__
void mul(size_t n, hipDoubleComplex *out, hipDoubleComplex *x, hipDoubleComplex *y)
{
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < n) out[i] = hipCmul(x[i], y[i]);
}

hipError_t cudaProduct(hipDoubleComplex *out, hipDoubleComplex * f2, hipDoubleComplex * f1, size_t n)
{
	saxpy <<< (n + 255) / 256, 256 >>>(n, out, f2, f1);

	return hipSuccess;
}

#else
hipError_t cudaProduct(hipDoubleComplex *out, hipDoubleComplex * f2, hipDoubleComplex * f1, size_t n)
{
	thrust::device_ptr<hipDoubleComplex> d_f1 = thrust::device_pointer_cast(f1);
	thrust::device_ptr<hipDoubleComplex> d_f1_e = d_f1 + n;
	thrust::device_ptr<hipDoubleComplex> d_f2 = thrust::device_pointer_cast(f2);
	thrust::device_ptr<hipDoubleComplex> d_out = thrust::device_pointer_cast(out);

	thrust::transform(d_f1, d_f1_e, d_f2, d_out, my_multiplies());
	return hipSuccess;
}
#endif


hipError_t cudaScale(hipDoubleComplex *out, hipDoubleComplex *in, double alpha, size_t n)
{
	thrust::device_ptr<hipDoubleComplex> d_in = thrust::device_pointer_cast(in);
	thrust::device_ptr<hipDoubleComplex> d_in_e = d_in + n;
	thrust::device_ptr<hipDoubleComplex> d_out = thrust::device_pointer_cast(out);

	thrust::transform(d_in, d_in_e, d_out, my_scale(alpha));
	return hipSuccess;
}
