#include "CudaUtility.h"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_ptr.h>

struct my_multiplies {
	typedef hipDoubleComplex first_argument_type;
	typedef hipDoubleComplex second_argument_type;
	typedef hipDoubleComplex result_type;
	__host__ __device__ hipDoubleComplex operator()(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) const
	{
		return hipCmul(lhs, rhs);
	}
};

struct myf_multiplies {
	typedef hipComplex first_argument_type;
	typedef hipComplex second_argument_type;
	typedef hipComplex result_type;
	__host__ __device__ hipComplex operator()(const hipComplex &lhs, const hipComplex &rhs) const
	{
		return hipCmulf(lhs, rhs);
	}
};

struct my_scale {
	typedef hipDoubleComplex first_argument_type;
	typedef hipDoubleComplex second_argument_type;
	typedef hipDoubleComplex result_type;

	my_scale(double m) : scale(m) { }
	double scale;
	__host__ __device__ hipDoubleComplex operator()(const hipDoubleComplex &lhs) const
	{
		return make_hipDoubleComplex(hipCreal(lhs)*scale, hipCimag(lhs)*scale);
	}
};

struct myf_scale {
	typedef hipComplex first_argument_type;
	typedef hipComplex second_argument_type;
	typedef hipComplex result_type;

	myf_scale(float m) : scale(m) {}
	float scale;
	__host__ __device__ hipComplex operator()(const hipComplex &lhs) const
	{
		return make_hipComplex(hipCrealf(lhs)*scale, hipCimagf(lhs)*scale);
	}
};

hipError_t cudaProduct(hipDoubleComplex * __restrict__ out, const hipDoubleComplex * __restrict__ f2, const hipDoubleComplex * __restrict__ f1, size_t n)
{
	thrust::device_ptr<const hipDoubleComplex> d_f1 = thrust::device_pointer_cast(f1);
	thrust::device_ptr<const hipDoubleComplex> d_f1_e = d_f1 + n;
	thrust::device_ptr<const hipDoubleComplex> d_f2 = thrust::device_pointer_cast(f2);
	thrust::device_ptr<hipDoubleComplex> d_out = thrust::device_pointer_cast(out);

	//thrust::transform(d_f1, d_f1_e, d_f2, d_out, my_multiplies());
	return hipSuccess;
}

hipError_t cudaProduct(hipComplex * __restrict__ out, const hipComplex * __restrict__ f2, const hipComplex * __restrict__ f1, size_t n)
{
	thrust::device_ptr<const hipComplex> d_f1 = thrust::device_pointer_cast(f1);
	thrust::device_ptr<const hipComplex> d_f1_e = d_f1 + n;
	thrust::device_ptr<const hipComplex> d_f2 = thrust::device_pointer_cast(f2);
	thrust::device_ptr<hipComplex> d_out = thrust::device_pointer_cast(out);

	thrust::transform(d_f1, d_f1_e, d_f2, d_out, myf_multiplies());
	return hipSuccess;
}


hipError_t cudaScale(hipDoubleComplex * __restrict__ out, const hipDoubleComplex * __restrict__ in, double alpha, size_t n)
{
	thrust::device_ptr<const hipDoubleComplex> d_in = thrust::device_pointer_cast(in);
	thrust::device_ptr<const hipDoubleComplex> d_in_e = d_in + n;
	thrust::device_ptr<hipDoubleComplex> d_out = thrust::device_pointer_cast(out);

	thrust::transform(d_in, d_in_e, d_out, my_scale(alpha));
	return hipSuccess;
}


hipError_t cudaScale(hipComplex * __restrict__ out, const hipComplex * __restrict__ in, float alpha, size_t n)
{
	thrust::device_ptr<const hipComplex> d_in = thrust::device_pointer_cast(in);
	thrust::device_ptr<const hipComplex> d_in_e = d_in + n;
	thrust::device_ptr<hipComplex> d_out = thrust::device_pointer_cast(out);

	thrust::transform(d_in, d_in_e, d_out, myf_scale(alpha));
	return hipSuccess;
}
